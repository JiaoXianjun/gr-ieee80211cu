#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>
#include <iostream>

int main(void)
{
    int N = 64;
    float x[64][64];
    float y[65536];
    float *d_x;

    hipMalloc(&d_x, N * N * sizeof(float));

    for (int i = 0; i < N; i++) {
        for(int j=0;j<N;j++)
        {
            x[i][j] = 2.0f;
        }
    }

    hipMemcpy(d_x, x, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(y, d_x, N * N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N*N; i++)
        maxError = max(maxError, abs(y[i] - 2.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
}