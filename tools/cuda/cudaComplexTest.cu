
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <iostream>
#include "hip/hip_complex.h"

__global__
void cuCompMultiKernel(int n, hipFloatComplex* x, hipFloatComplex* y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    y[i] = hipCmulf(x[i], y[i]);
  }
}

int main()
{
    std::cout<<"hipComplex Type Size: "<<sizeof(hipFloatComplex)<<std::endl;

    int n = 16;

    hipFloatComplex* signalX;
    hipFloatComplex* signalY;
    hipMalloc(&signalX, n*sizeof(hipFloatComplex));
    hipMalloc(&signalY, n*sizeof(hipFloatComplex));

    hipFloatComplex* x = (hipFloatComplex*)malloc(n*sizeof(hipFloatComplex));
    hipFloatComplex* y = (hipFloatComplex*)malloc(n*sizeof(hipFloatComplex));

    std::cout<<"original cuda complex type, real and imag of a hipFloatComplex"<<std::endl;
    for (int i = 0; i < n; i++) {
        x[i] = make_hipFloatComplex((float)i, (float)-i);
        std::cout <<hipCrealf(x[i]) << ", " << hipCimagf(x[i])<<std::endl;
        y[i] = make_hipFloatComplex((float)i, (float)i * 2.0f);
    }

    float* x2 = (float*) malloc(n*2*sizeof(float));
    float* y2 = (float*) malloc(n*2*sizeof(float));

    std::cout<<"convert hipFloatComplex to float array, real and imag"<<std::endl;
    for (int i = 0; i < n; i++) {
        x2[i*2] = (float)i;
        x2[i*2+1] = (float)-i;
        std::cout << x2[i*2] << ", " << x2[i*2+1] <<std::endl;
        y2[i*2] = (float)i;
        y2[i*2+1] = (float)i * 2.0f;
    }

    hipMemcpy(signalX, x, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(signalY, y, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    cuCompMultiKernel<<<(n+1024)/1024, 1024>>>(n, signalX, signalY);

    hipMemcpy(y, signalY, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    std::cout<<"cuda results"<<std::endl;
    for (int i = 0; i < n; i++) {
        std::cout <<hipCrealf(y[i]) << ", " << hipCimagf(y[i])<<std::endl;
    }

    hipMemcpy(signalX, (hipFloatComplex*)x2, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(signalY, (hipFloatComplex*)y2, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    cuCompMultiKernel<<<(n+1024)/1024, 1024>>>(n, signalX, signalY);

    hipMemcpy((hipFloatComplex*)y2, signalY, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    std::cout<<"cuda results 2"<<std::endl;
    for (int i = 0; i < n; i++) {
        std::cout << y2[i*2] << ", " << y2[i*2+1] <<std::endl;
    }

    free(x);
    free(y);
    free(x2);
    free(y2);

    hipFree(signalX);
    hipFree(signalY);
    return 0;
}